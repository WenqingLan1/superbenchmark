#include "hip/hip_runtime.h"
#include <nvbench/nvbench.cuh>

__global__ void empty_kernel() {}

void launch_bench(nvbench::state &state) {
  state.exec([](nvbench::launch &launch) {
    empty_kernel<<<1, 1, 0, launch.get_stream()>>>();
  });
}

NVBENCH_BENCH(launch_bench);